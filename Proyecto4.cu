#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <map>
#include <string>
#include <stdexcept>
#include <cmath>
#include <hip/hip_runtime.h>

using namespace std;

// Intervalos de 15mins en segundos
#define INTERVAL_SECONDS 900

// Categorias de aplicaciones de luz
#define OFFICE_MIN 300.0f
#define READING_MIN 500.0f
#define FACTORY_MIN 1000.0f
#define COUNTERPRODUCTIVE_MIN 5000.0f

// CUDA Kernel: Estadisticas por intervalo
__global__ void computeIntervalStats(
    const float *data, const int *intervals, int dataSize,
    float *tempSums, int *tempCounts,
    float *luxSums, float *luxSqSums, int *luxCounts)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < dataSize)
    {
        int intervalIdx = intervals[idx];

        printf("Hilo %d: Procesando en intervalo %d\n", idx, intervalIdx);

        // Actualización de estadisticas de intervalo
        atomicAdd(&tempSums[intervalIdx], data[idx * 3 + 1]);                      // Suma temperaturas
        atomicAdd(&tempCounts[intervalIdx], 1);                                    // Conteo temperaturas
        atomicAdd(&luxSums[intervalIdx], data[idx * 3 + 2]);                       // Suma lux
        atomicAdd(&luxSqSums[intervalIdx], data[idx * 3 + 2] * data[idx * 3 + 2]); // Suma cuadrados de lux (stdev)
        atomicAdd(&luxCounts[intervalIdx], 1);                                     // Conteo lux
    }
}

// CUDA Kernel: Calculo de Estadisticas Totales por Intervalo
__global__ void finalizeIntervalStats(
    const float *tempSums, const int *tempCounts, float *tempAverages,
    const float *luxSums, const float *luxSqSums, const int *luxCounts,
    float *luxStdDevs, int numIntervals)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < numIntervals)
    {

        // Promedio de Temperatura
        if (tempCounts[idx] > 0)
        {
            tempAverages[idx] = tempSums[idx] / tempCounts[idx];
        }

        // Desviación estándar de lux
        if (luxCounts[idx] > 0)
        {
            float mean = luxSums[idx] / luxCounts[idx];
            float meanSq = luxSqSums[idx] / luxCounts[idx];
            luxStdDevs[idx] = sqrtf(meanSq - (mean * mean));
        }

        // Mostrar que hilo maneja qué intervalo
        printf("Hilo %d: Finalizando intervalo %d\n", idx, idx);
    }
}

// CUDA Kernel: Tiempo pasado en cada categoria de iluminacion
__global__ void calculateLuxCategoryTime(
    const float *lux, const float *timeDeltas,
    float *timeOffice, float *timeReading, float *timeFactory, float *timeCounterproductive, int n)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n)
    {
        float deltaTime = timeDeltas[idx];

        // Agregar delta tiempo a cada categoría de iluminación
        if (lux[idx] >= OFFICE_MIN)
            atomicAdd(timeOffice, deltaTime);
        if (lux[idx] >= READING_MIN)
            atomicAdd(timeReading, deltaTime);
        if (lux[idx] >= FACTORY_MIN)
            atomicAdd(timeFactory, deltaTime);
        if (lux[idx] >= COUNTERPRODUCTIVE_MIN)
            atomicAdd(timeCounterproductive, deltaTime);

        // Mostrar que hilo maneja qué valor
        printf("Hilo %d: Procesando lux %.2f con delta tiempo %.2f\n", idx, lux[idx], deltaTime);
    }
}

// Parseo de timestamps a segundos
int parseTimestampToSeconds(const string &timestamp)
{
    int hours = 0, minutes = 0, seconds = 0;
    char delimiter;
    stringstream ss(timestamp);

    ss >> hours >> delimiter >> minutes >> delimiter >> seconds;
    if (ss.fail())
    {
        throw invalid_argument("Formato de timestamp inválido: " + timestamp);
    }

    return hours * 3600 + minutes * 60 + seconds;
}

// Lectura / parsing de CSV
vector<vector<float>> readCSV(const string &filename)
{
    ifstream file(filename);
    vector<vector<float>> data;
    string line;

    if (!file.is_open())
    {
        throw runtime_error("Error abriendo archivo: " + filename);
    }

    getline(file, line); // Saltar encabezado

    while (getline(file, line))
    {
        if (line.empty())
            continue;

        stringstream lineStream(line);
        string cell;
        vector<float> row;

        try
        {
            getline(lineStream, cell, ',');
            row.push_back(static_cast<float>(parseTimestampToSeconds(cell)));

            getline(lineStream, cell, ',');
            row.push_back(stof(cell));

            getline(lineStream, cell, ',');
            row.push_back(stof(cell));

            data.push_back(row);
        }
        catch (const invalid_argument &e)
        {
            cerr << "Error parseando línea: " << e.what() << endl;
        }
    }

    return data;
}

int main()
{
    const string filename = "data.csv";

    try
    {
        // Leer y parsear el archivo CSV
        auto data = readCSV(filename);

        if (data.empty())
        {
            cerr << "Error: No se encontraron datos en el archivo CSV." << endl;
            return -1;
        }

        // Preparar datos y calcular índices de intervalos
        int dataSize = data.size();
        vector<float> flatData(dataSize * 3);
        vector<int> intervalIndices(dataSize);

        int startTime = static_cast<int>(data[0][0]);
        for (int i = 0; i < dataSize; ++i)
        {
            flatData[i * 3] = data[i][0];
            flatData[i * 3 + 1] = data[i][1];
            flatData[i * 3 + 2] = data[i][2];
            intervalIndices[i] = static_cast<int>((data[i][0] - startTime) / INTERVAL_SECONDS);
        }

        int numIntervals = intervalIndices.back() + 1;

        // Asignar memoria en GPU
        float *d_data, *d_tempSums, *d_tempAverages, *d_luxSums, *d_luxSqSums, *d_luxStdDevs;
        float *d_timeOffice, *d_timeReading, *d_timeFactory, *d_timeCounterproductive;
        int *d_intervals, *d_tempCounts, *d_luxCounts;

        hipMalloc(&d_data, flatData.size() * sizeof(float));
        hipMalloc(&d_intervals, intervalIndices.size() * sizeof(int));
        hipMalloc(&d_tempSums, numIntervals * sizeof(float));
        hipMalloc(&d_tempAverages, numIntervals * sizeof(float));
        hipMalloc(&d_tempCounts, numIntervals * sizeof(int));
        hipMalloc(&d_luxSums, numIntervals * sizeof(float));
        hipMalloc(&d_luxSqSums, numIntervals * sizeof(float));
        hipMalloc(&d_luxStdDevs, numIntervals * sizeof(float));
        hipMalloc(&d_luxCounts, numIntervals * sizeof(int));
        hipMalloc(&d_timeOffice, sizeof(float));
        hipMalloc(&d_timeReading, sizeof(float));
        hipMalloc(&d_timeFactory, sizeof(float));
        hipMalloc(&d_timeCounterproductive, sizeof(float));

        // Inicializar memoria en GPU
        hipMemcpy(d_data, flatData.data(), flatData.size() * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_intervals, intervalIndices.data(), intervalIndices.size() * sizeof(int), hipMemcpyHostToDevice);
        hipMemset(d_tempSums, 0, numIntervals * sizeof(float));
        hipMemset(d_luxSums, 0, numIntervals * sizeof(float));
        hipMemset(d_luxSqSums, 0, numIntervals * sizeof(float));
        hipMemset(d_tempCounts, 0, numIntervals * sizeof(int));
        hipMemset(d_luxCounts, 0, numIntervals * sizeof(int));
        hipMemset(d_timeOffice, 0, sizeof(float));
        hipMemset(d_timeReading, 0, sizeof(float));
        hipMemset(d_timeFactory, 0, sizeof(float));
        hipMemset(d_timeCounterproductive, 0, sizeof(float));

        // Calcular estadísticas por intervalo
        int threads = 256;
        int blocks = (dataSize + threads - 1) / threads;
        computeIntervalStats<<<blocks, threads>>>(d_data, d_intervals, dataSize, d_tempSums, d_tempCounts, d_luxSums, d_luxSqSums, d_luxCounts);

        // Finalizar estadísticas por intervalo
        blocks = (numIntervals + threads - 1) / threads;
        finalizeIntervalStats<<<blocks, threads>>>(d_tempSums, d_tempCounts, d_tempAverages, d_luxSums, d_luxSqSums, d_luxCounts, d_luxStdDevs, numIntervals);

        // Preparar datos para calcular tiempo en categorías de iluminación
        vector<float> lux(dataSize), timeDeltas(dataSize);
        for (size_t i = 1; i < data.size(); ++i)
        {
            lux[i] = data[i][2];
            timeDeltas[i] = data[i][0] - data[i - 1][0];
        }

        float *d_lux, *d_timeDeltas;
        hipMalloc(&d_lux, lux.size() * sizeof(float));
        hipMalloc(&d_timeDeltas, timeDeltas.size() * sizeof(float));
        hipMemcpy(d_lux, lux.data(), lux.size() * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_timeDeltas, timeDeltas.data(), timeDeltas.size() * sizeof(float), hipMemcpyHostToDevice);

        // Calcular tiempos en categorías de iluminación
        blocks = (lux.size() + threads - 1) / threads;
        calculateLuxCategoryTime<<<blocks, threads>>>(d_lux, d_timeDeltas, d_timeOffice, d_timeReading, d_timeFactory, d_timeCounterproductive, lux.size());

        hipDeviceSynchronize();

        // Copiar resultados al host
        vector<float> tempAverages(numIntervals), luxStdDevs(numIntervals);
        float timeOffice, timeReading, timeFactory, timeCounterproductive;

        hipMemcpy(tempAverages.data(), d_tempAverages, numIntervals * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(luxStdDevs.data(), d_luxStdDevs, numIntervals * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(&timeOffice, d_timeOffice, sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(&timeReading, d_timeReading, sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(&timeFactory, d_timeFactory, sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(&timeCounterproductive, d_timeCounterproductive, sizeof(float), hipMemcpyDeviceToHost);

        // Imprimir resultados
        for (int i = 0; i < numIntervals; ++i)
        {
            cout << "Intervalo " << i << " (Tiempo: " << i * INTERVAL_SECONDS
                      << " - " << (i + 1) * INTERVAL_SECONDS << " segundos):\n";
            cout << "  Temperatura promedio: " << tempAverages[i] << "°C\n";
            cout << "  Desviación estándar de lux: " << luxStdDevs[i] << "\n";
        }

        cout << "\nTiempo en categorías de iluminación:\n";
        cout << "  Iluminación de oficina (300 lux+): " << timeOffice << " segundos\n";
        cout << "  Iluminación de lectura (500 lux+): " << timeReading << " segundos\n";
        cout << "  Iluminación de fábrica (1000 lux+): " << timeFactory << " segundos\n";
        cout << "  Iluminación contraproducente (5000 lux+): " << timeCounterproductive << " segundos\n";

        // Liberar memoria en GPU
        hipFree(d_data);
        hipFree(d_intervals);
        hipFree(d_tempSums);
        hipFree(d_tempCounts);
        hipFree(d_tempAverages);
        hipFree(d_luxSums);
        hipFree(d_luxSqSums);
        hipFree(d_luxCounts);
        hipFree(d_luxStdDevs);
        hipFree(d_timeOffice);
        hipFree(d_timeReading);
        hipFree(d_timeFactory);
        hipFree(d_timeCounterproductive);
        hipFree(d_lux);
        hipFree(d_timeDeltas);
    }
    catch (const exception &e)
    {
        cerr << "Error: " << e.what() << endl;
        return -1;
    }

    return 0;
}
